/*
* This implementation is the naive version of neural network classification.
* Here simple CUDA threads are launched which uses global memory for each access.
* No further optimization techniques are used in this version.
*/

#include <stdio.h>
#include <stdlib.h>

#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <float.h>

#define IMAGE_SIZE_TRAIN 60000
#define IMAGE_SIZE_TEST 10000
#define INPUT_SIZE 784
#define HIDDEN_SIZE 128
#define OUTPUT_SIZE 10
#define LEARNING_RATE 0.01
#define EPOCHS 3
#define BATCH_SIZE 64
#define NUM_CLASSES 10  // Digits 0-9

// Timer function
double get_time(clock_t start) {
    return (double)(clock() - start) / CLOCKS_PER_SEC;
}

// Allocate memory for a matrix (2D array as array of pointers)
double** allocateMatrix(int rows, int cols) {
    double** mat = (double**)malloc(rows * sizeof(double*));
    for (int i = 0; i < rows; i++) {
        mat[i] = (double*)malloc(cols * sizeof(double));
    }
    return mat;
}

// Free allocated matrix memory
void freeMatrix(double** mat, int rows) {
    for (int i = 0; i < rows; i++) {
        free(mat[i]);
    }
    free(mat);
}

void softmax(double* x, int size) {
    double sum = 0;
    for (int i = 0; i < size; i++) {
        x[i] = exp(x[i]);
        sum += x[i];
    }
    for (int i = 0; i < size; i++) {
        x[i] /= sum;
    }
}

// Helper to flatten a 2D matrix into a contiguous array.
double* flattenMatrix(double** matrix, int rows, int cols) {
    double* flat = (double*)malloc(rows * cols * sizeof(double));
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            flat[i * cols + j] = matrix[i][j];
        }
    }
    return flat;
}

// Neural network structure with additional device pointers.
typedef struct {
    double** W1;    // host 2D weight matrix for layer 1
    double** W2;    // host 2D weight matrix for layer 2
    double* b1;     // host biases for hidden layer
    double* b2;     // host biases for output layer

    // Flattened device arrays (row-major order)
    double* d_W1;
    double* d_W2;
    double* d_b1;
    double* d_b2;
} NeuralNetwork;

// Initialize neural network and allocate device memory.
NeuralNetwork* createNetwork() {
    NeuralNetwork* net = (NeuralNetwork*)malloc(sizeof(NeuralNetwork));
    net->W1 = allocateMatrix(HIDDEN_SIZE, INPUT_SIZE);
    net->W2 = allocateMatrix(OUTPUT_SIZE, HIDDEN_SIZE);
    net->b1 = (double*)calloc(HIDDEN_SIZE, sizeof(double));
    net->b2 = (double*)calloc(OUTPUT_SIZE, sizeof(double));

    srand(time(NULL));
    for (int i = 0; i < HIDDEN_SIZE; i++)
        for (int j = 0; j < INPUT_SIZE; j++)
            net->W1[i][j] = ((double)rand() / RAND_MAX) * 0.01;

    for (int i = 0; i < OUTPUT_SIZE; i++)
        for (int j = 0; j < HIDDEN_SIZE; j++)
            net->W2[i][j] = ((double)rand() / RAND_MAX) * 0.01;

    // Flatten weight matrices.
    double* h_W1 = flattenMatrix(net->W1, HIDDEN_SIZE, INPUT_SIZE);
    double* h_W2 = flattenMatrix(net->W2, OUTPUT_SIZE, HIDDEN_SIZE);

    // Allocate device memory for weights and biases.
    hipMalloc(&(net->d_W1), HIDDEN_SIZE * INPUT_SIZE * sizeof(double));
    hipMalloc(&(net->d_W2), OUTPUT_SIZE * HIDDEN_SIZE * sizeof(double));
    hipMalloc(&(net->d_b1), HIDDEN_SIZE * sizeof(double));
    hipMalloc(&(net->d_b2), OUTPUT_SIZE * sizeof(double));

    // Copy from host to device.
    hipMemcpy(net->d_W1, h_W1, HIDDEN_SIZE * INPUT_SIZE * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(net->d_W2, h_W2, OUTPUT_SIZE * HIDDEN_SIZE * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(net->d_b1, net->b1, HIDDEN_SIZE * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(net->d_b2, net->b2, OUTPUT_SIZE * sizeof(double), hipMemcpyHostToDevice);

    free(h_W1);
    free(h_W2);

    return net;
}

// Kernel to compute hidden layer and apply ReLU activation directly.
__global__ void forward_hidden_kernel(const double *W1, const double *B1, const double *input, double *hidden) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (index >= HIDDEN_SIZE)  return;
    
    double sum = B1[index];
    for (int imagecol = 0; imagecol < INPUT_SIZE; imagecol++) {
        sum += W1[index * INPUT_SIZE + imagecol] * input[imagecol];
    }

    hidden[index] = (sum > 0) ? sum : 0;
}

// Kernel to compute output layer (logits).
__global__ void forward_output_kernel(const double *W2, const double *b2, const double *hidden, double *output) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= OUTPUT_SIZE)  return;
    
    double sum = b2[index];
    for (int hiddencol = 0; hiddencol < HIDDEN_SIZE; hiddencol++) {
        sum += W2[index * HIDDEN_SIZE + hiddencol] * hidden[hiddencol];
    }

    output[index] = sum;
}

// Optimized forward pass using the GPU.
void forward(NeuralNetwork* net, double* input, double* hidden, double* output)  {
    double *d_input, *d_hidden, *d_output;
    hipMalloc(&d_input, INPUT_SIZE * sizeof(double));
    hipMalloc(&d_hidden, HIDDEN_SIZE * sizeof(double));
    hipMalloc(&d_output, OUTPUT_SIZE * sizeof(double));

    // Copy the numerical image into GPU variable.
    hipMemcpy(d_input, input, INPUT_SIZE * sizeof(double), hipMemcpyHostToDevice);

    int THREADSPERBLOCK = -1, BLOCKSIZE = -1;

    // Calculates the hidden layer weights, and applys ReLU activation.
    THREADSPERBLOCK = 32;
    BLOCKSIZE = (HIDDEN_SIZE + THREADSPERBLOCK - 1) / THREADSPERBLOCK;
    forward_hidden_kernel<<<BLOCKSIZE, THREADSPERBLOCK>>>(net->d_W1, net->d_b1, d_input, d_hidden);
    hipDeviceSynchronize();

    // Calculates the output layer weights, using hidden weights.
    THREADSPERBLOCK = 16;
    BLOCKSIZE = (OUTPUT_SIZE + THREADSPERBLOCK - 1) / THREADSPERBLOCK;
    forward_output_kernel<<<BLOCKSIZE, THREADSPERBLOCK>>>(net->d_W2, net->d_b2, d_hidden, d_output);
    hipDeviceSynchronize();

    // Copy back data from the Device.
    hipMemcpy(hidden, d_hidden, HIDDEN_SIZE * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(output, d_output, OUTPUT_SIZE * sizeof(double), hipMemcpyDeviceToHost);

    // Apply softmax on the output.
    softmax(output, OUTPUT_SIZE);

    hipFree(d_input);
    hipFree(d_hidden);
    hipFree(d_output);
}

// Kernel to compute hidden layer gradient for backpropagation.
__global__ void compute_d_hidden(double* d_hidden, const double* W2, const double* d_output, const double* hidden) {
    int index = threadIdx.x;
    if (index >= HIDDEN_SIZE)  return;
    
    double gradient = 0;
    for (int outputcol = 0; outputcol < OUTPUT_SIZE; outputcol++) {
        gradient += W2[outputcol * HIDDEN_SIZE + index] * d_output[outputcol];
    }
    d_hidden[index] = gradient * (hidden[index] > 0);
}

// Kernel to update output layer weights.
__global__ void updateW2(double* W2, const double* d_output, const double* hidden, double learning_rate) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;  // output index
    int col = blockIdx.y * blockDim.y + threadIdx.y;  // hidden index
    if (row < OUTPUT_SIZE && col < HIDDEN_SIZE) {
        int idx = row * HIDDEN_SIZE + col;
        W2[idx] -= learning_rate * d_output[row] * hidden[col];
    }
}

// Kernel to update hidden layer weights.
__global__ void updateW1(double* W1, const double* d_hidden, const double* input, double learning_rate) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;  // hidden index
    int col = blockIdx.y * blockDim.y + threadIdx.y;    // input index
    if (row < HIDDEN_SIZE && col < INPUT_SIZE) {
        int idx = row * INPUT_SIZE + col;
        W1[idx] -= learning_rate * d_hidden[row] * input[col];
    }
}

// Backward pass: compute gradients and update device weights.
void backward(NeuralNetwork* net, double* input, double* hidden, double* output, double* target) {
    double d_output[OUTPUT_SIZE], d_hidden[HIDDEN_SIZE];
    
    // Compute gradient for output layer.
    for (int i = 0; i < OUTPUT_SIZE; i++) {
        d_output[i] = output[i] - target[i];
    }

    // Allocate device memory for input and intermediate gradients.
    double *d_input, *d_hidden_d, *d_output_d;
    hipMalloc(&d_input, INPUT_SIZE * sizeof(double));
    hipMalloc(&d_hidden_d, HIDDEN_SIZE * sizeof(double));
    hipMalloc(&d_output_d, OUTPUT_SIZE * sizeof(double));

    // Copy input and output gradients to device.
    hipMemcpy(d_input, input, INPUT_SIZE * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_output_d, d_output, OUTPUT_SIZE * sizeof(double), hipMemcpyHostToDevice);
    
    // Copy hidden from the FORWARD() function.
    double *d_hidden_forward;
    hipMalloc(&d_hidden_forward, HIDDEN_SIZE * sizeof(double));
    hipMemcpy(d_hidden_forward, hidden, HIDDEN_SIZE * sizeof(double), hipMemcpyHostToDevice);

    // Compute hidden layer gradient on device.
    int THREADSPERBLOCK = 128;  
    int BLOCKSIZE = (HIDDEN_SIZE + THREADSPERBLOCK - 1) / THREADSPERBLOCK;
    compute_d_hidden<<<BLOCKSIZE, THREADSPERBLOCK>>>(d_hidden_d, net->d_W2, d_output_d, d_hidden_forward);
    hipDeviceSynchronize();

    // Copy the computed hidden gradient back to host.
    hipMemcpy(d_hidden, d_hidden_d, HIDDEN_SIZE * sizeof(double), hipMemcpyDeviceToHost);

    // Define grid and block dimensions for weight update kernels.
    dim3 blockDim(16, 16);
    dim3 gridDim_W2((OUTPUT_SIZE + blockDim.x - 1) / blockDim.x, (HIDDEN_SIZE + blockDim.y - 1) / blockDim.y);
    updateW2<<<gridDim_W2, blockDim>>>(net->d_W2, d_output_d, d_hidden_forward, LEARNING_RATE);
    hipDeviceSynchronize();

    dim3 gridDim_W1((HIDDEN_SIZE + blockDim.x - 1) / blockDim.x, (INPUT_SIZE + blockDim.y - 1) / blockDim.y);
    updateW1<<<gridDim_W1, blockDim>>>(net->d_W1, d_hidden_d, d_input, LEARNING_RATE);
    hipDeviceSynchronize();

    // Update biases on host and copy to device.
    for (int i = 0; i < OUTPUT_SIZE; i++) {
        net->b2[i] -= LEARNING_RATE * d_output[i];
    }

    for (int i = 0; i < HIDDEN_SIZE; i++) {
        net->b1[i] -= LEARNING_RATE * d_hidden[i];
    }

    // Copy back to host to save data.
    hipMemcpy(net->d_b2, net->b2, OUTPUT_SIZE * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(net->d_b1, net->b1, HIDDEN_SIZE * sizeof(double), hipMemcpyHostToDevice);

    // Free allocated device memory.
    hipFree(d_input);
    hipFree(d_hidden_d);
    hipFree(d_output_d);
    hipFree(d_hidden_forward);
}

// Train the network on all training images.
void train(NeuralNetwork* net, double** images, double** labels, int numImages) {
    clock_t total_start = clock();
    for (int epoch = 0; epoch < EPOCHS; epoch++) {
        clock_t epoch_start = clock();
        double loss = 0.0;
        int correct = 0;
        for (int i = 0; i < numImages; i++) {
            double hidden[HIDDEN_SIZE], output[OUTPUT_SIZE];
            forward(net, images[i], hidden, output);
            backward(net, images[i], hidden, output, labels[i]);

            // Compute cross-entropy loss and accuracy.
            for (int k = 0; k < OUTPUT_SIZE; k++) {
                loss -= labels[i][k] * log(output[k]);
            }
            int pred = 0, actual = 0;
            for (int j = 0; j < OUTPUT_SIZE; j++) {
                if (output[j] > output[pred])
                    pred = j;
                if (labels[i][j] > labels[i][actual])
                    actual = j;
            }
            if (pred == actual)
                correct++;
        }
        printf("Epoch %d - Loss: %.4f - Train Accuracy: %.2f%% - Time: %.3fs\n",
               epoch + 1, loss / numImages, (correct / (double)numImages) * 100, get_time(epoch_start));
    }
    printf("Total training time: %.3fs\n", get_time(total_start));
}

// Evaluate accuracy on test images.
void evaluate(NeuralNetwork* net, double** images, double** labels, int numImages) {
    int correct = 0;
    for (int i = 0; i < numImages; i++) {
        double hidden[HIDDEN_SIZE], output[OUTPUT_SIZE];
        forward(net, images[i], hidden, output);
        int pred = 0, actual = 0;
        for (int j = 0; j < OUTPUT_SIZE; j++) {
            if (output[j] > output[pred])
                pred = j;
            if (labels[i][j] > labels[i][actual])
                actual = j;
        }
        if (pred == actual)
            correct++;
    }
    printf("Test Accuracy: %.2f%%\n", (correct / (double)numImages) * 100);
}

// Read MNIST images.
double** loadMNISTImages(const char* filename, int numImages) {
    FILE* file = fopen(filename, "rb");
    if (!file) {
        printf("Error opening %s\n", filename);
        exit(1);
    }
    fseek(file, 16, SEEK_SET);
    double** images = allocateMatrix(numImages, INPUT_SIZE);
    for (int i = 0; i < numImages; i++) {
        for (int j = 0; j < INPUT_SIZE; j++) {
            unsigned char pixel;
            if (fread(&pixel, sizeof(unsigned char), 1, file) != 1) {
                fprintf(stderr, "Error: Failed to read pixel\n");
                fclose(file);
                exit(EXIT_FAILURE);
            }
            images[i][j] = pixel / 255.0;
        }
    }

    fclose(file);
    return images;
}

// Read MNIST labels.
double** loadMNISTLabels(const char* filename, int numLabels) {
    FILE* file = fopen(filename, "rb");
    if (!file) {
        printf("Error opening %s\n", filename);
        exit(1);
    }
    fseek(file, 8, SEEK_SET);
    double** labels = allocateMatrix(numLabels, OUTPUT_SIZE);
    for (int i = 0; i < numLabels; i++) {
        unsigned char label;
        if (fread(&label, sizeof(unsigned char), 1, file) != 1) {
            fprintf(stderr, "Error: Failed to read label\n");
            fclose(file);
            exit(EXIT_FAILURE);
        }

        for (int j = 0; j < OUTPUT_SIZE; j++) {
            labels[i][j] = (j == label) ? 1.0 : 0.0;
        }
    }

    fclose(file);
    return labels;
}

// Free network memory, including device memory.
void freeNetwork(NeuralNetwork* net) {
    freeMatrix(net->W1, HIDDEN_SIZE);
    freeMatrix(net->W2, OUTPUT_SIZE);
    free(net->b1);
    free(net->b2);
    free(net);
    
    hipFree(net->d_W1);
    hipFree(net->d_W2);
    hipFree(net->d_b1);
    hipFree(net->d_b2);
}

// Main function.
int main() {
    printf("MNIST Neural Network\n\n");
    double** train_images = loadMNISTImages("data/train-images.idx3-ubyte", IMAGE_SIZE_TRAIN);
    double** train_labels = loadMNISTLabels("data/train-labels.idx1-ubyte", IMAGE_SIZE_TRAIN);
    double** test_images = loadMNISTImages("data/t10k-images.idx3-ubyte", IMAGE_SIZE_TEST);
    double** test_labels = loadMNISTLabels("data/t10k-labels.idx1-ubyte", IMAGE_SIZE_TEST);

    NeuralNetwork* net = createNetwork();
    train(net, train_images, train_labels, IMAGE_SIZE_TRAIN);
    evaluate(net, test_images, test_labels, IMAGE_SIZE_TEST);

    freeNetwork(net);
    return 0;
}